#include "hip/hip_runtime.h"
#include "numc.cuh"

void* Managed::operator new(size_t len) {
#ifdef DEBUG
    printf("Managed new\n");
#endif
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
}

void Managed::operator delete(void *ptr) {
#ifdef DEBUG
    printf("Managed delete\n");
#endif
    hipDeviceSynchronize();
    hipFree(ptr);
}

template <typename Op, typename T>
__host__
void apply(const Matrix<T> &x, const Matrix<T> &y, Matrix<T> &dest, Op op) {
    dim3 threadsPerBlock(THREADS_PER_BLOCK_2D, THREADS_PER_BLOCK_2D);
    dim3 numBlocks(
        (dest.getXDim() + THREADS_PER_BLOCK_2D - 1) / THREADS_PER_BLOCK_2D, 
        (dest.getYDim() + THREADS_PER_BLOCK_2D - 1) / THREADS_PER_BLOCK_2D
    );
    operation_kernel<<<numBlocks, threadsPerBlock>>>(*x.matrixGPU, *y.matrixGPU,
            *dest.matrixGPU, op);
    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess){
        printf("Sync kernel error: %d %s\n", errSync, hipGetErrorString(errSync));
    }
    if (errAsync != hipSuccess){
        printf("Async kernel error: %d %s\n", errAsync, hipGetErrorString(errAsync));
    }
}

template<typename Op, typename T>
__global__
void operation_kernel(const MatrixGPU<T> &x, const MatrixGPU<T> &y, MatrixGPU<T> &dest, Op op) {
    size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t xStride = gridDim.x * blockDim.x;
    size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    size_t yStride = gridDim.y * blockDim.y;
    for (size_t i = xIndex; i < dest.getXDim(); i+= xStride){
        for (size_t j = yIndex; j < dest.getYDim(); j+= yStride){
            dest(i, j) = op(x(i, j), y(i, j));
        }
    }
}

template class Matrix<float>;
INSTANTIATE_operation_kernel(float, Add);

#include "hip/hip_runtime.h"
#include "numc.cuh"


template <typename Op, typename T>
__host__
void apply(const Matrix<T> &x, const Matrix<T> &y, Matrix<T> &dest, Op op) {
    dim3 threadsPerBlock(THREADS_PER_BLOCK_2D, THREADS_PER_BLOCK_2D);
    dim3 numBlocks(
        (dest.getXDim() + THREADS_PER_BLOCK_2D - 1) / THREADS_PER_BLOCK_2D, 
        (dest.getYDim() + THREADS_PER_BLOCK_2D - 1) / THREADS_PER_BLOCK_2D
    );
    operation_kernel<<<numBlocks, threadsPerBlock>>>(
            x.matrixGPU, y.matrixGPU, dest.matrixGPU, op);
    hipError_t errSync  = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess){
        printf("Sync kernel error: %d %s\n", errSync, hipGetErrorString(errSync));
    }
    if (errAsync != hipSuccess){
        printf("Async kernel error: %d %s\n", errAsync, hipGetErrorString(errAsync));
    }
}


template<typename Op, typename T>
__global__
void operation_kernel(const typename Matrix<T>::MatrixGPU *xptr,
                      const typename Matrix<T>::MatrixGPU *yptr, 
                      typename Matrix<T>::MatrixGPU *destptr, 
                      Op op) {
    const MatrixGPU &x = *xptr; 
    const MatrixGPU &y = *yptr; 
    MatrixGPU &dest = *destptr; 
    size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t xStride = gridDim.x * blockDim.x;
    size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    size_t yStride = gridDim.y * blockDim.y;
    for (size_t i = xIndex; i < dest.getXDim(); i+= xStride){
        for (size_t j = yIndex; j < dest.getYDim(); j+= yStride){
            dest(i, j) = op(x(i, j), y(i, j));
        }
    }
}

template class Matrix<float>;
template void operation_kernel<Add, float>;
